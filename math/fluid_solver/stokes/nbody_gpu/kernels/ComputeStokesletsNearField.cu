#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "ComputeStokesletsNearFieldKernel.cu"


	/**
	 * @brief ...
	 *
	 * @param h_targets ...
	 * @param h_velocities ...
	 * @param h_sources ...
	 * @param h_forces ...
	 * @param deltas ...
	 * @param num_sources ...
	 * @param num_targets ...
	 * @param num_clusters ...
	 * 
	 **/
	void ComputeStokesletsNearField( const float *h_targets, float *h_velocities, const float * h_sources, const float * h_forces, float delta, int num_sources, int num_targets, int num_clusters ) 
	{
        int size_targets = 3 * num_targets;
        int size_sources = 3 * num_sources;
        float *d_velocities, *d_targets, *d_sources, *d_strengths;
        hipMalloc((void**)&d_velocities, sizeof( float) * size_targets);
        hipMalloc((void**)&d_targets,    sizeof( float) * size_targets);
        hipMalloc((void**)&d_sources,    sizeof( float) * size_sources);
        hipMalloc((void**)&d_strengths,  sizeof( float) * size_sources);
        hipMemcpy(d_velocities, h_velocities, size_targets*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_targets,    h_targets,    size_targets*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_sources,    h_sources,    size_sources*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_strengths,  h_forces,  size_sources*sizeof(float), hipMemcpyHostToDevice);

        int block_threads = num_sources/num_clusters;
        int num_blocks = num_sources / block_threads + (num_sources % block_threads == 0 ? 0 : 1);
        int sharedMemSize = 2 * block_threads * sizeof(float3);
        dim3 threads(block_threads, 1, 1);
        dim3 grid(num_blocks, 1, 1);
        ComputeStokesletsNearFieldKernel<<<grid,threads,sharedMemSize>>>(
			(float3 *)d_targets, 
			(float3 *)d_velocities, 
			(float3 *)d_sources, 
			(float3 *)d_strengths, 
			delta, 
			num_targets
		);
		//Copying data from host to device
        hipMemcpy(h_velocities,  d_velocities,  size_targets*sizeof(float), hipMemcpyDeviceToHost);
        hipFree((void**)d_velocities);
        hipFree((void**)d_targets);
        hipFree((void**)d_sources);
        hipFree((void**)d_strengths);
	}

