#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "ComputeImagesNearFieldKernel.cu"

 void ComputeImagesNearField( const float *h_targets, float *h_velocities, const float * h_sources, const float * h_forces, float deltas, int num_sources, int num_targets, int num_clusters )
    {

        int size_targets = 3* num_targets;
        int size_sources = 3* num_sources;
        float *d_velocities, *d_targets, *d_sources, *d_forces;
        hipMalloc((void**)&d_velocities, sizeof( float) * size_targets);
        hipMalloc((void**)&d_targets, sizeof( float) * size_targets);
        hipMalloc((void**)&d_sources, sizeof( float) * size_sources);
        hipMalloc((void**)&d_forces, sizeof( float) * size_sources);
        hipMemcpy(d_velocities, h_velocities, size_targets*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_targets, h_targets, size_targets*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_sources, h_sources, size_sources*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_forces, h_forces, size_sources*sizeof(float), hipMemcpyHostToDevice);

        int block_threads = num_sources/num_clusters;
        int num_blocks 	  = num_clusters;
        int sharedMemSize = 2 * block_threads * sizeof(float3);
        dim3 threads(block_threads, 1, 1);
        dim3 grid(num_blocks, 1, 1);
        ComputeImagesNearFieldKernel<<< grid,threads,sharedMemSize >>> (
                (float3*)d_targets,
                (float3*)d_velocities,
                (float3*)d_sources,
                (float3*)d_forces,
                deltas,
                num_sources,
                num_targets
        );

        //Copying data from host to device
        hipMemcpy(h_velocities, 	d_velocities, 	size_targets*sizeof(float), hipMemcpyDeviceToHost);

        hipFree((void**)d_velocities);
        hipFree((void**)d_targets);
        hipFree((void**)d_sources);
        hipFree((void**)d_forces);

    }






