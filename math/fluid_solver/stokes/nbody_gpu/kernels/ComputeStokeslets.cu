#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include "../gpu_compute_velocity.hpp"
#include "ComputeStokesletsKernel.cu"

/**
 * @brief ...
 *
 * @param h_targets ...
 * @param h_sources ...
 * @param h_strengths ...
 * @param h_velocities ...
 * @param deltas ...
 **/
template<>
void ComputeStokeslets<float>(const float *h_targets, float *h_velocities, const float * h_sources, const float * h_strengths, float delta, size_t num_sources, size_t num_targets, bool with_images)
{
    hipSetDeviceFlags(hipDeviceMapHost);
    // First, allocate space on device
    size_t size_targets = 3 * num_targets;
    size_t size_sources = 3 * num_sources;
    float *d_velocities, *d_targets, *d_sources, *d_strengths;
    hipHostRegister(const_cast<float*>(h_targets), sizeof(float) * size_targets, hipHostRegisterMapped);
    hipHostRegister(h_velocities, sizeof(float) * size_targets, hipHostRegisterMapped);
    hipHostRegister(const_cast<float*>(h_sources), sizeof(float) * size_sources, hipHostRegisterMapped);
    hipHostRegister(const_cast<float*>(h_strengths), sizeof(float) * size_sources, hipHostRegisterMapped);

    hipHostGetDevicePointer((void **) & d_targets, (void *)h_targets, 0);
    hipHostGetDevicePointer((void **) & d_velocities, (void *)h_velocities, 0);
    hipHostGetDevicePointer((void **) & d_sources, (void *)h_sources, 0);
    hipHostGetDevicePointer((void **) & d_strengths, (void *)h_strengths, 0);

    size_t block_threads = 32;
    size_t num_blocks = num_targets / block_threads + (num_targets % block_threads == 0 ? 0 : 1);
    size_t sharedMemSize = 2 * block_threads * sizeof(float3);
    dim3 threads(block_threads, 1, 1);
    dim3 grid(num_blocks, 1, 1);
    ComputeStokesletsKernel<<<grid,threads,sharedMemSize>>>(
        (float3 *)d_targets,
        (float3 *)d_velocities,
        (float3 *)d_sources,
        (float3 *)d_strengths,
        delta,
        num_sources,
        num_targets, with_images
    );

    //Copying data from host to device
    hipHostUnregister(const_cast<float*>(h_targets));
    hipHostUnregister(h_velocities);
    hipHostUnregister(const_cast<float*>(h_sources));
    hipHostUnregister(const_cast<float*>(h_strengths));

}






